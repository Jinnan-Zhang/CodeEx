#include "hip/hip_runtime.h"
/*
* @compile: nvcc -o HelloWorld.cu.o HelloWorld.cu
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <HelloWorld.h>
#define N (2048 * 2048)
#define THREAD_PER_BLOCK 512
#define BLOCKS_NUM (N / THREAD_PER_BLOCK)



__global__ void helloworld(void)
{
    printf("Hello World!\n");
}
void random_ints(int *a, int n)
{
    for (int i = 0; i < n; i++)
        *(a + i) = rand() % 10;
}
__global__ void mul(int *a, int *b, int *sub_sum)
{
    __shared__ int c[THREAD_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[threadIdx.x] = a[index] * b[index];
    __syncthreads();
    for (int i = THREAD_PER_BLOCK / 2; i > 0; i = i / 2)
    {
        if (threadIdx.x < i)
        {
            c[threadIdx.x] += (c[threadIdx.x + i]);
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
        sub_sum[blockIdx.x] = c[threadIdx.x];
}

void DoVector()
{
    int *a, *b, *sub_sum, sum;  //host copies
    int *d_a, *d_b, *d_sub_sum; //device copies
    int size = N * sizeof(int);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_sub_sum, sizeof(int) * BLOCKS_NUM);

    a = (int *)malloc(size);
    random_ints(a, N);
    b = (int *)malloc(size);
    random_ints(b, N);
    sub_sum = (int *)malloc(sizeof(int) * BLOCKS_NUM);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    StartTimeChrono(1);
    mul<<<BLOCKS_NUM, THREAD_PER_BLOCK>>>(d_a, d_b, d_sub_sum);

    StopTimeChrono(1);
    hipMemcpy(sub_sum, d_sub_sum, sizeof(int) * BLOCKS_NUM, hipMemcpyDeviceToHost);
    sum = 0;
    for (int i = 0; i < BLOCKS_NUM; i++)
        sum += sub_sum[i];
    StartTimeChrono(2);
    int sum_cpu = 0;
    for (int i = 0; i < N; i++)
    {
        // printf("%d\t%d\n", a[i], b[i]);
        sum_cpu += (a[i] * b[i]);
    }
    StopTimeChrono(2);
    printf(" GPU result:%d\t CPU result:%d\n", sum, sum_cpu);
    PrintTimeChrono(1, "GPU");
    PrintTimeChrono(2, "CPU");
    free(a), free(b), free(sub_sum);
    hipFree(d_a), hipFree(d_b), hipFree(d_sub_sum);
    hipDeviceReset();
}

int main(void)
{
    // helloworld<<<1,32>>>();
    // hipDeviceReset();
    DoVector();

    return 0;
}