#include "hip/hip_runtime.h"
#include "particle.h"
#include <stdlib.h>
#include <stdio.h>
#include "tcu.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <iostream>

//try thrust
void TryThrust(int argc, char **argv)
{
	thrust::device_vector<int> X(10);
	thrust::device_vector<int> Y(10);
	thrust::device_vector<int> Z(10);

	thrust::sequence(X.begin(),X.end());
	thrust::transform(X.begin(),X.end(),Y.begin(),thrust::negate<int>());
	thrust::fill(Z.begin(),Z.end(),2);
	thrust::copy(Y.begin(),Y.end(),std::ostream_iterator<int>(std::cout,"\n"));
}

__global__ void advanceParticles(float dt, particle *pArray, int nParticles)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < nParticles)
	{
		pArray[idx].advance(dt);
	}
}

void DoParticle(int argc, char **argv)
{
	hipError_t error;
	int n = 1000000;
	if (argc > 1)
	{
		n = atoi(argv[1]);
	} // Number of particles
	if (argc > 2)
	{
		srand(atoi(argv[2]));
	} // Random seed

	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("0 %s\n", hipGetErrorString(error));
		exit(1);
	}

	particle *pArray = new particle[n];
	particle *devPArray = NULL;
	hipMalloc(&devPArray, n * sizeof(particle));
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("1 %s\n", hipGetErrorString(error));
		exit(1);
	}

	hipMemcpy(devPArray, pArray, n * sizeof(particle), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("2 %s\n", hipGetErrorString(error));
		exit(1);
	}

	for (int i = 0; i < 100; i++)
	{
		float dt = (float)rand() / (float)RAND_MAX; // Random distance each step
		advanceParticles<<<1 + n / 256, 256>>>(dt, devPArray, n);
		error = hipGetLastError();
		if (error != hipSuccess)
		{
			printf("3 %s\n", hipGetErrorString(error));
			exit(1);
		}

		hipDeviceSynchronize();
	}
	hipMemcpy(pArray, devPArray, n * sizeof(particle), hipMemcpyDeviceToHost);

	v3 totalDistance(0, 0, 0);
	v3 temp;
	for (int i = 0; i < n; i++)
	{
		temp = pArray[i].getTotalDistance();
		totalDistance.x += temp.x;
		totalDistance.y += temp.y;
		totalDistance.z += temp.z;
	}
	float avgX = totalDistance.x / (float)n;
	float avgY = totalDistance.y / (float)n;
	float avgZ = totalDistance.z / (float)n;
	float avgNorm = sqrt(avgX * avgX + avgY * avgY + avgZ * avgZ);
	printf("Moved %d particles 100 steps. Average distance traveled is |(%f, %f, %f)| = %f\n",
		   n, avgX, avgY, avgZ, avgNorm);
}