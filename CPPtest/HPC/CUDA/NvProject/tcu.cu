#include "hip/hip_runtime.h"
#include "particle.h"
#include <stdlib.h>
#include <stdio.h>

__global__ void advanceParticles(float dt, particle * pArray, int nParticles)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		pArray[idx].advance(dt);
	}
}