/*
* @compile: nvcc -o HelloWorld.cu.o HelloWorld.cu
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#define N 64

__global__ void helloworld(void)
{
    printf("Hello World!\n");
}
void random_ints(int *a, int n)
{
    for (int i = 0; i < n; i++)
        *(a + i) = rand() % 10;
}
__global__ void mul(int *a, int *b, int *sum)
{
    __shared__ int c[N];
    c[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];
    __syncthreads();
    for (int i = N / 2; i > 0; i = i / 2)
    {
        if (threadIdx.x < i)
        {
            c[threadIdx.x] += (c[threadIdx.x + i]);
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
        *sum = c[threadIdx.x];
}

void DoVector()
{
    int *a, *b, sum;        //host copies
    int *d_a, *d_b, *d_sum; //device copies
    int size = N * sizeof(int);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_sum, sizeof(int));

    a = (int *)malloc(size);
    random_ints(a, N);
    b = (int *)malloc(size);
    random_ints(b, N);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    mul<<<1, N>>>(d_a, d_b, d_sum);

    hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
    int sum_cpu = 0;
    for (int i = 0; i < N; i++)
    {
        // printf("%d\t%d\n", a[i], b[i]);
        sum_cpu += (a[i] * b[i]);
    }
    printf(" GPU result:%d\t CPU result:%d\n", sum, sum_cpu);
    free(a), free(b);
    hipFree(d_a), hipFree(d_b), hipFree(d_sum);
}

int main(void)
{
    // helloworld<<<1,32>>>();
    // cudaDeviceReset();
    DoVector();

    return 0;
}