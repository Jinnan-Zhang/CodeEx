#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloworld(void)
{
    printf("Hello World!\n");
}

int main(void)
{
    helloworld<<<1,1>>>();
    hipDeviceReset();
    return 0;
}