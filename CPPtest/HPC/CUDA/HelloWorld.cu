/*
* @compile: nvcc -o HelloWorld.cu.o HelloWorld.cu
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#define N 48

#pragma once
/*
usage:
StartTimeChrono(num);
...
StopTimeChrono(num);
PrintTimeChrono(num,"discription")
*/
#include <chrono>

#define StartTimeChrono(_num) \
    auto start##_num = std::chrono::high_resolution_clock::now()

#define StopTimeChrono(_num) \
    auto stop##_num = std::chrono::high_resolution_clock::now()

#ifdef _WIN32
#define PrintTimeChrono(_num, _str)                                                                    \
    auto dura##_num = std::chrono::duration_cast<std::chrono::microseconds>(stop##_num - start##_num); \
    printf("Time for calculating %s is %f millisecond.\n", _str, dura##_num.count())
#else
#define PrintTimeChrono(_num, _str)                                                                    \
    auto dura##_num = std::chrono::duration_cast<std::chrono::microseconds>(stop##_num - start##_num); \
    printf("Time for calculating %s is %.8f second.\n", _str, dura##_num.count() / 1.e6)
#endif

__global__ void helloworld(void)
{
    printf("Hello World!\n");
}
void random_ints(int *a, int n)
{
    for (int i = 0; i < n; i++)
        *(a + i) = rand() % 10;
}
__global__ void mul(int *a, int *b, int *sum)
{
    __shared__ int c[N];
    c[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];
    __syncthreads();
    for (int i = N / 2; i > 0; i = i / 2)
    {
        if (threadIdx.x < i)
        {
            c[threadIdx.x] += (c[threadIdx.x + i]);
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
        *sum = c[threadIdx.x];
}

void DoVector()
{
    int *a, *b, sum;        //host copies
    int *d_a, *d_b, *d_sum; //device copies
    int size = N * sizeof(int);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_sum, sizeof(int));

    a = (int *)malloc(size);
    random_ints(a, N);
    b = (int *)malloc(size);
    random_ints(b, N);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    StartTimeChrono(1);
    mul<<<1, N>>>(d_a, d_b, d_sum);

    StopTimeChrono(1);
    hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
    StartTimeChrono(2);
    int sum_cpu = 0;
    for (int i = 0; i < N; i++)
    {
        // printf("%d\t%d\n", a[i], b[i]);
        sum_cpu += (a[i] * b[i]);
    }
    StopTimeChrono(2);
    printf(" GPU result:%d\t CPU result:%d\n", sum, sum_cpu);
    PrintTimeChrono(1, "GPU");
    PrintTimeChrono(2, "CPU");
    free(a), free(b);
    hipFree(d_a), hipFree(d_b), hipFree(d_sum);
    hipDeviceReset();

}

int main(void)
{
    // helloworld<<<1,32>>>();
    // cudaDeviceReset();
    DoVector();

    return 0;
}